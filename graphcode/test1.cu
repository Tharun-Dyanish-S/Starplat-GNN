#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

class GraphNeuralNetworks{
	public:
		int32_t num_features;
		double *weights;
		double *bias;
		double epsilon = 0.01;
		double grad_epsilon;
		double *aggregatedFeatures;
		double *preActivatedFeatures;
		double *postActivatedFeatures;
		double *grad_pre_act_output;
		double *grad_weights;
		double *grad_bias;
		double *m_weights;
		double *m_biases;
		double m_epsilon = 0.0f;
		double v_epsilon = 0.0f;
		double *v_weights;
		double *v_biases;

		//Constructor to initialize layers
		__device__ GraphNeuralNetworks() : num_features(0), weights(nullptr), bias(nullptr), epsilon(0.01), grad_epsilon(0.0), aggregatedFeatures(nullptr), preActivatedFeatures(nullptr), postActivatedFeatures(nullptr),grad_pre_act_output(nullptr), grad_weights(nullptr), grad_bias(nullptr), m_weights(nullptr), m_biases(nullptr), m_epsilon(0.0f), v_epsilon(0.0f), v_weights(nullptr), v_biases(nullptr) {}


		__device__ void initializeLayers(int32_t layer_num, int32_t num_feat_current, int32_t num_feat_next){
			weights = new double[num_feat_current + num_feat_next];
			bias = new double[num_feat_next];
			grad_weights = new double[num_feat_current + num_feat_next];
			grad_bias = new double[num_feat_next];
			m_weights = new double[num_feat_current + num_feat_next];
			m_biases = new double[num_feat_next];	
			v_weights = new double[num_feat_current + num_feat_next];
		}
};

__global__ void initializeLayers_cuda(GraphNeuralNetworks *layers, int32_t currentSize, int32_t nextSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	layers[idx].initializeLayers(idx, currentSize, nextSize);
}

void initializeLayers_cuda(vector<int> numnodespreLayer, char *transformation){
	int num_layers = numNodespreLayer.size() - 1;
	GraphNeuralNetworks *layers;
	if(strcmp(transformation, "xaviers") == 0){
		transformType = 1;
	}
	else if(strcmp(transformation, "he") == 0){
		transformType = 2;
	}
	hipMallocManaged(&layers, num_layers * sizeof(GraphNeuralNetworks));
	for(int i = 1;i<num_layers-1;i++){
	initializeLayers_cuda<<<1, 1>>>(layers, numNodesperLayer[i], numNodesperLayer[i+1]);
	}
	hipDeviceSynchronize();

	

}

void forwardProp_cuda(int32_t layer){
	GraphNeuralNetworks *layers;
	aggregatedGCN_cuda<<<numNodes, numFeatures>>>(layers, layer);
	hipDeviceSynchronize();
	GCN_NN<<<numNodes, numFeatures>>>(layers, layer);
	hipDeviceSynchronize();
}