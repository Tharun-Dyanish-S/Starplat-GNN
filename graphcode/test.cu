#include <iostream>
#include <hip/hip_runtime.h>

class MyClass {
public:
    int32_t numEyes;        // Integer data member
    double* numMistakes;    // Pointer to a dynamically allocated array

    // Constructor to initialize numEyes and prepare memory for numMistakes
    __device__ MyClass() : numEyes(0), numMistakes(nullptr) {}

    // Method to initialize the object on the device
    __device__ void initialize(int32_t eyes, int32_t array_size) {
        numEyes = eyes;
        numMistakes = new double[array_size];  // Allocate memory on the device
        for (int i = 0; i < array_size; i++) {
            numMistakes[i] = static_cast<double>(i) * 1.1;  // Initialize the array
        }
    }

    // Method to free dynamically allocated memory
    __device__ void freeArray() {
        delete[] numMistakes;  // Free the dynamically allocated array
    }

    // Print method for debugging (prints the data)
    __device__ void printData(int32_t array_size) {
        printf("numEyes: %d\n", numEyes);
        for (int i = 0; i < array_size; i++) {
            printf("numMistakes[%d] = %f\n", i, numMistakes[i]);
        }
    }
};

// Kernel to initialize an array of MyClass objects and print their values
__global__ void initializeAndProcess(MyClass* myClassArray, int32_t array_size, int32_t num_classes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
        // Initialize each MyClass object in the array
        myClassArray[idx].initialize(idx, array_size);  // Pass a unique numEyes and array size

        // Print the values for debugging (executed on the GPU)
        for(int i = 0;i<array_size;i++){
            printf("numMistakes[%d] = %f,  numEyes  = %d\n", i, myClassArray[idx].numMistakes[i],myClassArray[idx] .numEyes);
        }
    }


// Kernel to clean up the dynamically allocated arrays in MyClass objects
__global__ void freeMemory(MyClass* myClassArray, int32_t num_classes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_classes) {
        myClassArray[idx].freeArray();  // Free each object's array
    }
}

int main() {
    const int32_t num_classes = 3;    // Number of MyClass objects
    const int32_t array_size = 5;     // Size of each numMistakes array

    // Allocate memory for an array of MyClass objects on the device
    MyClass* d_myClassArray;
    hipMalloc(&d_myClassArray, num_classes * sizeof(MyClass));

    // Launch kernel to initialize the objects and their arrays
    initializeAndProcess<<<1, num_classes>>>(d_myClassArray, array_size, num_classes);
    hipDeviceSynchronize();

    // Launch kernel to free dynamically allocated memory in each object
    freeMemory<<<1, num_classes>>>(d_myClassArray, num_classes);
    hipDeviceSynchronize();

    // Free the device memory allocated for the array of objects


    return 0;
}
