// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "test.h"

void yjhjnj(graph& g,int k)

{
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp;
    temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp;
    temp = g.edgeList[i];
    h_data[i] = temp;
  }

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);

  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V > threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  thrust::device_vector<char>  s = "Hello";
  thrust::device_vector<char>  s1;
  s1 = "World";
  message_passs("STRRR",1);


  yjhjnj_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data);
  hipDeviceSynchronize();




  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
