// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "GCN_test.h"

void GCN(graph& g,GNN  gnn,thrust::device_vector<int> neuronsPerHiddenLayer)

{
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();




  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V > threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  initializeLayers_cuda(neuronsPerHiddenLayer,"xaviers");

  int num_epoch = 1;
  bool x = true;
  while(x){
    if (num_epoch == 100){ // if filter begin 
      x = false;

    } // if filter end
    num_epoch = num_epoch + 1;
    int layerr = 0;
    bool y = 0;
    while(y){
      layerr = layerr + 1;
      forwardPass_cuda(layerr);



    }while(layerr){
      layerr--;
      backPropogation_cuda(layerr);



    }

  }
  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
